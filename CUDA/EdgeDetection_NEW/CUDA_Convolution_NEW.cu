#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "BMP.h"
#include "Sobel.h"
#include "EdgeDetection_Defines.h"

// IMPORTANT: THE SOBEL OPERATOR NEEDS A GRAYSCALE IMAGE
// Convert image to grayscale:       https://www.grayscaleimage.com/#downloadtoolgray

// TO DO: DEFINE THESE FUNCTIONS
void GPU_AllocateMemory(void);
void GPU_CopyData(void);
void GPU_KernelLaunch_Convolution(void);
void GPU_KernelLaunch_SobelOperation(void);
void GPU_KernelLaunch_SobelFindMax(void);
void GPU_KernelLaunch_SobelNormalization(void);


// TO DO: COPY TO THE SOBEL INSTANCE BUFFERS INSTEAD OF THESE SEPERATE GLOBAL BUFFERS
int16_t h_output_conv_x[CONV_OUT_ROW_MAX_LEN * CONV_OUT_COL_MAX_LEN];
int16_t h_output_conv_y[CONV_OUT_ROW_MAX_LEN * CONV_OUT_COL_MAX_LEN]; 
uint16_t h_output_sobel[SOBEL_OUT_ROW_MAX_LEN * SOBEL_OUT_COL_MAX_LEN]; 
uint8_t h_output_sobel_norm[SOBEL_OUT_ROW_MAX_LEN * SOBEL_OUT_COL_MAX_LEN]; 

// CLI example invoke command: <application_filepath(.exe)> <input_bmp_image_path(.bmp)> <output_bmp_image_path(.bmp)>
int main(int argc, char *argv[])
{
    BMP bmp;
    char* filepath_edgedetection_in; 
	char* filepath_edgedetection_out;
	
	// Check if the user entered a filepath for the input and output image on invoking the application.
	if (argc != 3) 
	{
        printf("Usage: %s <input_bmp_image_path> <output_bmp_image_path>\n", argv[0]);
        return 1;
    }
	
	// Input and Output filepaths
	filepath_edgedetection_in  = argv[1];
	filepath_edgedetection_out = argv[2];

    // Read the input image dimensions and 1D image data + convert to 2D image data.
    // InitInputImage(&bmp, filepath_edgedetection_in);
    BMP_Init(&bmp, filepath_edgedetection_in);
    // TEST WRITE OF THE ORIGINAL INPUT IMAGE => WORKS BUT THE JETSON FAILS TO DISPLAY THIS BMP FORMAT WHILE MY WINDOWS LAPTOP DISPLAYS IT JUST FINE
    // BMP_SaveGrayscaleImage(filepath_edgedetection_out, bmp.grayscale_data, bmp.width, bmp.height); 

    Sobel_Initialize(&sobel, bmp.width, bmp.height);
    printf("Edge detection output width: %d, height: %d\r\n", sobel.width, sobel.height);

    // Pointer variables to keep track of GPU allocated memory
    uint8_t *d_image; 
	int16_t *d_output_conv_x; 
    int16_t *d_output_conv_y; 
    uint16_t *d_output_sobel; 
    uint8_t *d_output_sobel_norm; 

    // Allocate memory on the GPU for the input image data, the x and y edge detection convolution results.
    hipMalloc((void **)&d_image, BMP_IMAGE_ROW_MAX_SIZE * BMP_IMAGE_COL_MAX_SIZE * sizeof(uint8_t));
	hipMalloc((void **)&d_output_conv_x, CONV_OUT_ROW_MAX_LEN * CONV_OUT_COL_MAX_LEN * sizeof(int16_t)); 
	hipMalloc((void **)&d_output_conv_y, CONV_OUT_ROW_MAX_LEN * CONV_OUT_COL_MAX_LEN * sizeof(int16_t)); 
    hipMalloc((void **)&d_output_sobel, SOBEL_OUT_ROW_MAX_LEN * SOBEL_OUT_COL_MAX_LEN * sizeof(uint16_t)); 
    hipMalloc((void **)&d_output_sobel_norm, SOBEL_OUT_ROW_MAX_LEN * SOBEL_OUT_COL_MAX_LEN * sizeof(uint8_t)); 

    // Copy the input image and sobel x and y filter kernel data to the GPU
    hipMemcpy(d_image, bmp.grayscale_data, sizeof(bmp.grayscale_data), hipMemcpyHostToDevice); // Using the 1D data => The conversion from the original 1D to 2D image data can be skipped!
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_x), sobel_kernel_x, sizeof(sobel_kernel_x));
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_y), sobel_kernel_y, sizeof(sobel_kernel_y));

    // Launch the convolution kernels:
    dim3 blockSize_conv(16, 16);
    dim3 gridSize_conv((sobel.width + blockSize_conv.x - 1) / blockSize_conv.x, 
                  (sobel.height + blockSize_conv.y - 1) / blockSize_conv.y);
    Kernel_SobelConvolution_X<<<gridSize_conv, blockSize_conv>>>(d_image, d_output_conv_x, sobel.width, sobel.height); 
    Kernel_SobelConvolution_Y<<<gridSize_conv, blockSize_conv>>>(d_image, d_output_conv_y, sobel.width, sobel.height); 
    // Wait for the convolution kernels to complete.
    hipDeviceSynchronize();
    // Copy the GPU computated results back to the CPU
    hipMemcpy(h_output_conv_x, d_output_conv_x, sizeof(h_output_conv_x), hipMemcpyDeviceToHost);
    hipMemcpy(h_output_conv_y, d_output_conv_y, sizeof(h_output_conv_y), hipMemcpyDeviceToHost);
    
    // Launch the sobel operation kernel:
    dim3 blockSize_sobel(16, 16); 
    dim3 gridSize_sobel((sobel.width + blockSize_sobel.x - 1) / blockSize_sobel.x, 
                  (sobel.height + blockSize_sobel.y - 1) / blockSize_sobel.y);
    Kernel_SobelMagnitude<<<gridSize_sobel, blockSize_sobel>>>(d_output_conv_x, d_output_conv_y, d_output_sobel, sobel.width, sobel.height); 
    // Wait for the sobel operation kernel to complete.
    hipDeviceSynchronize();
    // Copy the GPU computated result back to the CPU
    hipMemcpy(h_output_sobel, d_output_sobel, sizeof(h_output_sobel), hipMemcpyDeviceToHost);

    // Launch the kernel to find the maximum value of the sobel output
    int sobel_size = sobel.width * sobel.height;
    // Allocate on device
    uint32_t* d_max_val;
    hipMalloc(&d_max_val, sizeof(uint16_t));
    hipMemset(d_max_val, 0, sizeof(uint16_t));
    // Find max
    int threads_find_max = 256;
    int blocks_find_max = (sobel_size + threads_find_max - 1) / threads_find_max;
    Kernel_FindMaxValue<<<blocks_find_max, threads_find_max>>>(d_output_sobel, d_max_val, sobel_size);
    // Wait for the sobel operation kernel to complete.
    hipDeviceSynchronize();
    // Copy max value to host (optional, or pass directly)
    uint16_t h_max_val;
    hipMemcpy(&h_max_val, d_max_val, sizeof(uint16_t), hipMemcpyDeviceToHost);
    // printf("Max sobel value: %d\r\n", h_max_val); 

    /* 
    // C sequential version to find the max value
    uint16_t sobel_max = 0;
    for(int32_t i = 0; i < sobel_size; ++i)
    {
        sobel_max = (sobel_max > *(h_output_sobel + i)) ? sobel_max : *(h_output_sobel + i);
    }
    printf("Max sobel value: %d\r\n", sobel_max); // BUG: ALSO PRINTS 0
    */

    // Launch the sobel output normalization kernel:
    dim3 blockSize(16, 16);
    dim3 gridSize((sobel.width + 15) / 16, (sobel.height + 15) / 16);
    Kernel_NormalizeSobelOutput<<<gridSize, blockSize>>>(d_output_sobel, d_output_sobel_norm, h_max_val, sobel.width, sobel.height);
    // Wait for the sobel operation kernel to complete.
    hipDeviceSynchronize();
    // Copy the GPU computated result back to the CPU
    hipMemcpy(h_output_sobel_norm, d_output_sobel_norm, sizeof(h_output_sobel_norm), hipMemcpyDeviceToHost);

    BMP_SaveGrayscaleImage(filepath_edgedetection_out, h_output_sobel_norm, sobel.width, sobel.height);

    // Cleanup
    hipFree(d_image);
    hipFree(d_output_conv_x);
    hipFree(d_output_conv_y);
    hipFree(d_output_sobel);
    hipFree(d_output_sobel_norm);
    hipFree(d_max_val);

    return 0;
}

void GPU_AllocateMemory(void)
{
    
}

void GPU_CopyData(void)
{

}

void GPU_KernelLaunch_Convolution(void)
{
    
}

void GPU_KernelLaunch_SobelOperation(void)
{
    
}

void GPU_KernelLaunch_SobelFindMax(void)
{
    
}

void GPU_KernelLaunch_SobelNormalization(void)
{

}
