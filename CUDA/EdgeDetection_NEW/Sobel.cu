#include "hip/hip_runtime.h"
#include "Sobel.h"

__constant__ int8_t d_kernel_x[KERNEL_ROW_LEN][KERNEL_COL_LEN]; 
__constant__ int8_t d_kernel_y[KERNEL_ROW_LEN][KERNEL_COL_LEN];

__global__ void Kernel_SobelConvolution_X(uint8_t *d_image, int16_t *d_output, uint16_t output_width, uint16_t output_height)
{
	// Calculate the (row, col) position of the thread's output pixel.
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < output_height && col < output_width)
    {
        int16_t sum = 0;
        for (int i = 0; i < KERNEL_ROW_LEN; ++i)
        {
            for (int j = 0; j < KERNEL_COL_LEN; ++j)
            {
                int img_row = row + i;
                int img_col = col + j;
                sum += d_kernel_x[i][j] * d_image[img_row * (output_width + 2) + img_col]; // Image width: (output_width + 2)
            }
        }
        d_output[row * output_width + col] = sum;
    }
}

__global__ void Kernel_SobelConvolution_Y(uint8_t *d_image, int16_t *d_output, uint16_t output_width, uint16_t output_height)
{
	// Calculate the (row, col) position of the thread's output pixel.
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < output_height && col < output_width)
    {
        int16_t sum = 0;
        for (int i = 0; i < KERNEL_ROW_LEN; ++i)
        {
            for (int j = 0; j < KERNEL_COL_LEN; ++j)
            {
                int img_row = row + i;
                int img_col = col + j;
                sum += d_kernel_y[i][j] * d_image[img_row * (output_width + 2) + img_col]; // Image width: (output_width + 2)
            }
        }
        d_output[row * output_width + col] = sum;
    }
}

__global__ void Kernel_SobelMagnitude(int16_t* d_convolution_output_x, int16_t* d_convolution_output_y, uint16_t* d_sobel_output, uint16_t width, uint16_t height)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width)
    {
        int idx = row * width + col;

        int gx = d_convolution_output_x[idx];
        int gy = d_convolution_output_y[idx];
        d_sobel_output[idx] = abs(gx) + abs(gy);
    }
}

__global__ void Kernel_FindMaxValue(uint16_t* d_input, uint32_t* d_max_val, uint16_t size)
{
    __shared__ uint32_t local_max[256];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    local_max[tid] = 0;

    if (idx < size)
        local_max[tid] = d_input[idx];

    __syncthreads();

    // Reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
            local_max[tid] = max(local_max[tid], local_max[tid + stride]);
        __syncthreads();
    }

    // First thread in block writes its result to global memory
    if (tid == 0)
        atomicMax(d_max_val, local_max[0]);
}

__global__ void Kernel_NormalizeSobelOutput(uint16_t* d_sobel_output, uint8_t* d_sobel_norm_output, uint16_t max_sobel_val, int width, int height)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width)
    {
        int idx = row * width + col;
        d_sobel_norm_output[idx] = (d_sobel_output[idx] * 255) / max(max_sobel_val, 1);
    }
}
