#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "BMP.h"
#include "Sobel.h"
#include "EdgeDetection_Defines.h"

// IMPORTANT: THE SOBEL OPERATOR NEEDS A GRAYSCALE IMAGE
// Convert image to grayscale:       https://www.grayscaleimage.com/#downloadtoolgray

typedef struct GPU_MemoryPointers
{
    uint8_t *d_image; 
	int16_t *d_output_conv_x; 
    int16_t *d_output_conv_y; 
    uint16_t *d_output_sobel; 
    uint32_t* d_sobel_max_val;
    uint8_t *d_output_sobel_norm; 
} GPU_MemoryPointers;

// TO DO: DEFINE THESE FUNCTIONS
void CPU_InitInputImage(int argc, char *argv[], BMP *bmp, char** input_image_path, char** output_image_path);
void CPU_InitSobel(Sobel *sobel, BMP *bmp);

void GPU_AllocateMemory(GPU_MemoryPointers *d_memory_pointers);
void GPU_FreeMemory(GPU_MemoryPointers *d_memory_pointers);
void GPU_CopyImageData(GPU_MemoryPointers *d_memory_pointers, BMP *bmp);
void GPU_KernelLaunch_Convolution(Sobel* sobel, GPU_MemoryPointers *d_memory_pointers, int16_t* h_output_conv_x, int16_t* h_output_conv_y);
void GPU_KernelLaunch_SobelOperation(Sobel* sobel, GPU_MemoryPointers *d_memory_pointers, uint16_t* h_output_sobel);
void GPU_KernelLaunch_SobelFindMax(Sobel* sobel, GPU_MemoryPointers *d_memory_pointers, uint16_t* h_max_val);
void GPU_KernelLaunch_SobelNormalization(Sobel* sobel, GPU_MemoryPointers *d_memory_pointers, uint8_t* h_output_sobel_norm, uint16_t h_max_val);

// TO DO: COPY TO THE SOBEL INSTANCE BUFFERS INSTEAD OF THESE SEPERATE GLOBAL BUFFERS
int16_t h_output_conv_x[CONV_OUT_ROW_MAX_LEN * CONV_OUT_COL_MAX_LEN];
int16_t h_output_conv_y[CONV_OUT_ROW_MAX_LEN * CONV_OUT_COL_MAX_LEN]; 
uint16_t h_output_sobel[SOBEL_OUT_ROW_MAX_LEN * SOBEL_OUT_COL_MAX_LEN]; 
uint16_t h_max_val = 0;
uint8_t h_output_sobel_norm[SOBEL_OUT_ROW_MAX_LEN * SOBEL_OUT_COL_MAX_LEN]; 

// CLI example invoke command: <application_filepath(.exe)> <input_bmp_image_path(.bmp)> <output_bmp_image_path(.bmp)>
int main(int argc, char *argv[])
{
    BMP bmp;
    char* filepath_edgedetection_in; 
	char* filepath_edgedetection_out;
	
    CPU_InitInputImage(argc, argv, &bmp, &filepath_edgedetection_in, &filepath_edgedetection_out);
    CPU_InitSobel(&sobel, &bmp);


    GPU_MemoryPointers d_memory_pointers;
    GPU_AllocateMemory(&d_memory_pointers);
    GPU_CopyImageData(&d_memory_pointers, &bmp);

    GPU_KernelLaunch_Convolution(&sobel, &d_memory_pointers, h_output_conv_x, h_output_conv_y); // TO DO: CHANGE HOST MEMORY TO SOBEL INSTANCE WHEN THOSE BUFFERS ARE USED.
    GPU_KernelLaunch_SobelOperation(&sobel, &d_memory_pointers, h_output_sobel);
    GPU_KernelLaunch_SobelFindMax(&sobel, &d_memory_pointers, &h_max_val);
    GPU_KernelLaunch_SobelNormalization(&sobel, &d_memory_pointers, h_output_sobel_norm, h_max_val);

    // THE JETSON FAILS TO DISPLAY THIS BMP FORMAT WITH A COUPLE IMAGE VIEWERS THAT WERE TESTED WHILE MY WINDOWS LAPTOP DISPLAYS IT JUST FINE
    BMP_SaveGrayscaleImage(filepath_edgedetection_out, h_output_sobel_norm, sobel.width, sobel.height);

    GPU_FreeMemory(&d_memory_pointers);

    return 0;
}

void CPU_InitInputImage(int argc, char *argv[], BMP *bmp, char** input_image_path, char** output_image_path)
{
    // Check if the user entered a filepath for the input and output image on invoking the application.
	if (argc != 3) 
	{
        printf("Usage: %s <input_bmp_image_path> <output_bmp_image_path>\n", argv[0]);
        // exit 1;
    }
	
	// Input and Output filepaths
	*input_image_path = argv[1];
	*output_image_path = argv[2];

    // Read the input image dimensions and 1D image data + convert to 2D image data.
    BMP_Init(bmp, *input_image_path);
    // TEST WRITE OF THE ORIGINAL INPUT IMAGE => WORKS BUT THE JETSON FAILS TO DISPLAY THIS BMP FORMAT WHILE MY WINDOWS LAPTOP DISPLAYS IT JUST FINE
    // BMP_SaveGrayscaleImage(output_image_path, bmp->grayscale_data, bmp->width, bmp->height); 
}

void CPU_InitSobel(Sobel *sobel, BMP *bmp)
{
    Sobel_Initialize(sobel, bmp->width, bmp->height);
    printf("Edge detection output width: %d, height: %d\r\n", sobel->width, sobel->height);
}

void GPU_AllocateMemory(GPU_MemoryPointers *d_memory_pointers)
{
    hipMalloc((void **)&(d_memory_pointers->d_image), BMP_IMAGE_ROW_MAX_SIZE * BMP_IMAGE_COL_MAX_SIZE * sizeof(uint8_t));
	hipMalloc((void **)&(d_memory_pointers->d_output_conv_x), CONV_OUT_ROW_MAX_LEN * CONV_OUT_COL_MAX_LEN * sizeof(int16_t)); 
	hipMalloc((void **)&(d_memory_pointers->d_output_conv_y), CONV_OUT_ROW_MAX_LEN * CONV_OUT_COL_MAX_LEN * sizeof(int16_t)); 
    hipMalloc((void **)&(d_memory_pointers->d_output_sobel), SOBEL_OUT_ROW_MAX_LEN * SOBEL_OUT_COL_MAX_LEN * sizeof(uint16_t)); 
    hipMalloc((void **)&(d_memory_pointers->d_sobel_max_val), sizeof(uint16_t));
    hipMemset(d_memory_pointers->d_sobel_max_val, 0, sizeof(uint16_t));
    hipMalloc((void **)&(d_memory_pointers->d_output_sobel_norm), SOBEL_OUT_ROW_MAX_LEN * SOBEL_OUT_COL_MAX_LEN * sizeof(uint8_t));
}

void GPU_CopyImageData(GPU_MemoryPointers *d_memory_pointers, BMP *bmp)
{
    hipMemcpy(d_memory_pointers->d_image, bmp->grayscale_data, sizeof(bmp->grayscale_data), hipMemcpyHostToDevice); // Using the 1D data => The conversion from the original 1D to 2D image data can be skipped!
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_x), sobel_kernel_x, sizeof(sobel_kernel_x));
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_y), sobel_kernel_y, sizeof(sobel_kernel_y));
}

void GPU_KernelLaunch_Convolution(Sobel* sobel, GPU_MemoryPointers *d_memory_pointers, int16_t* h_output_conv_x, int16_t* h_output_conv_y)
{
    // Launch the convolution kernels:
    dim3 blockSize_conv(16, 16);
    dim3 gridSize_conv((sobel->width + blockSize_conv.x - 1) / blockSize_conv.x, (sobel->height + blockSize_conv.y - 1) / blockSize_conv.y);
    Kernel_SobelConvolution_X<<<gridSize_conv, blockSize_conv>>>(d_memory_pointers->d_image, d_memory_pointers->d_output_conv_x, sobel->width, sobel->height); 
    Kernel_SobelConvolution_Y<<<gridSize_conv, blockSize_conv>>>(d_memory_pointers->d_image, d_memory_pointers->d_output_conv_y, sobel->width, sobel->height); 
    // Wait for the convolution kernels to complete.
    hipDeviceSynchronize();
    // Copy the GPU computated results back to the CPU
    hipMemcpy(h_output_conv_x, d_memory_pointers->d_output_conv_x, CONV_OUT_ROW_MAX_LEN * CONV_OUT_COL_MAX_LEN * sizeof(int16_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_output_conv_y, d_memory_pointers->d_output_conv_y, CONV_OUT_ROW_MAX_LEN * CONV_OUT_COL_MAX_LEN * sizeof(int16_t), hipMemcpyDeviceToHost);
}

void GPU_KernelLaunch_SobelOperation(Sobel* sobel, GPU_MemoryPointers *d_memory_pointers, uint16_t* h_output_sobel)
{
    dim3 blockSize_sobel(16, 16); 
    dim3 gridSize_sobel((sobel->width + blockSize_sobel.x - 1) / blockSize_sobel.x, (sobel->height + blockSize_sobel.y - 1) / blockSize_sobel.y);
    Kernel_SobelMagnitude<<<gridSize_sobel, blockSize_sobel>>>(d_memory_pointers->d_output_conv_x, d_memory_pointers->d_output_conv_y, d_memory_pointers->d_output_sobel, sobel->width, sobel->height); 
    // Wait for the sobel operation kernel to complete.
    hipDeviceSynchronize();
    // Copy the GPU computated result back to the CPU
    hipMemcpy(h_output_sobel, d_memory_pointers->d_output_sobel, SOBEL_OUT_ROW_MAX_LEN * SOBEL_OUT_COL_MAX_LEN * sizeof(uint16_t), hipMemcpyDeviceToHost);
}

void GPU_KernelLaunch_SobelFindMax(Sobel* sobel, GPU_MemoryPointers *d_memory_pointers, uint16_t* h_max_val)
{
    int sobel_size = sobel->width * sobel->height;
    // Allocate on device
    // uint32_t* d_max_val;
    // // hipMalloc(&d_max_val, sizeof(uint16_t));
    // hipMalloc(&(d_memory_pointers->d_sobel_max_val), sizeof(uint16_t));
    // // hipMemset(d_max_val, 0, sizeof(uint16_t));
    // hipMemset(d_memory_pointers->d_sobel_max_val, 0, sizeof(uint16_t));
    // Find max
    int threads_find_max = 256;
    int blocks_find_max = (sobel_size + threads_find_max - 1) / threads_find_max;
    // Kernel_FindMaxValue<<<blocks_find_max, threads_find_max>>>(d_output_sobel, d_max_val, sobel_size);
    Kernel_FindMaxValue<<<blocks_find_max, threads_find_max>>>(d_memory_pointers->d_output_sobel, d_memory_pointers->d_sobel_max_val, sobel_size);
    // Wait for the sobel operation kernel to complete.
    hipDeviceSynchronize();
    // Copy max value to host (optional, or pass directly)
    // uint16_t h_max_val;
    // hipMemcpy(&h_max_val, d_max_val, sizeof(uint16_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_max_val, d_memory_pointers->d_sobel_max_val, sizeof(uint16_t), hipMemcpyDeviceToHost);
    // printf("Max sobel value: %d\r\n", h_max_val); 

    /* 
    // C sequential version to find the max value
    uint16_t sobel_max = 0;
    for(int32_t i = 0; i < sobel_size; ++i)
    {
        sobel_max = (sobel_max > *(h_output_sobel + i)) ? sobel_max : *(h_output_sobel + i);
    }
    printf("Max sobel value: %d\r\n", sobel_max); // BUG: ALSO PRINTS 0
    */
}

void GPU_KernelLaunch_SobelNormalization(Sobel* sobel, GPU_MemoryPointers *d_memory_pointers, uint8_t* h_output_sobel_norm, uint16_t h_max_val)
{
    dim3 blockSize(16, 16);
    dim3 gridSize((sobel->width + 15) / 16, (sobel->height + 15) / 16);
    Kernel_NormalizeSobelOutput<<<gridSize, blockSize>>>(d_memory_pointers->d_output_sobel, d_memory_pointers->d_output_sobel_norm, h_max_val, sobel->width, sobel->height);
    // Wait for the sobel operation kernel to complete.
    hipDeviceSynchronize();
    // Copy the GPU computated result back to the CPU
    hipMemcpy(h_output_sobel_norm, d_memory_pointers->d_output_sobel_norm, SOBEL_OUT_ROW_MAX_LEN * SOBEL_OUT_COL_MAX_LEN * sizeof(uint8_t), hipMemcpyDeviceToHost);
}

void GPU_FreeMemory(GPU_MemoryPointers *d_memory_pointers)
{
    hipFree(d_memory_pointers->d_image);
    hipFree(d_memory_pointers->d_output_conv_x);
    hipFree(d_memory_pointers->d_output_conv_y);
    hipFree(d_memory_pointers->d_output_sobel);
    hipFree(d_memory_pointers->d_sobel_max_val);
    hipFree(d_memory_pointers->d_output_sobel_norm);
}
