#include "hip/hip_runtime.h"
#include "Pooling.h"

// CUDA kernels: parallelize the 2D pooling over thread blocks. 
// Each thread will handle one 2x2 region in the input image, and write to one output element.

__global__ void Kernel_PoolingMax(const uint8_t* input, uint8_t* output, int input_width, int input_height) 
{
    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;

    int out_width = input_width / 2; // TO DO: check if >> 1, which equals /2, is faster. Also try different compiler optimatization flags.
    int out_height = input_height / 2;

    if (out_x < out_width && out_y < out_height) 
    {
        int in_x = out_x * 2; // TO DO: check if << 1, which equals *2, is faster. Also try different compiler optimatization flags.
        int in_y = out_y * 2;

        int idx00 = in_y * input_width + in_x;
        int idx01 = idx00 + 1;
        int idx10 = idx00 + input_width;
        int idx11 = idx10 + 1;

        uint8_t max_val = input[idx00];
        max_val = max(input[idx01], max_val);
        max_val = max(input[idx10], max_val);
        max_val = max(input[idx11], max_val);

        output[out_y * out_width + out_x] = max_val;
    }
}

__global__ void Kernel_PoolingMin(const uint8_t* input, uint8_t* output, int input_width, int input_height) 
{
    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;

    int out_width = input_width / 2; // TO DO: check if >> 1, which equals /2, is faster. Also try different compiler optimatization flags.
    int out_height = input_height / 2;

    if (out_x < out_width && out_y < out_height) 
    {
        int in_x = out_x * 2; // TO DO: check if << 1, which equals *2, is faster. Also try different compiler optimatization flags.
        int in_y = out_y * 2;

        int idx00 = in_y * input_width + in_x;
        int idx01 = idx00 + 1;
        int idx10 = idx00 + input_width;
        int idx11 = idx10 + 1;

        uint8_t min_val = input[idx00];
        min_val = min(input[idx01], min_val);
        min_val = min(input[idx10], min_val);
        min_val = min(input[idx11], min_val);

        output[out_y * out_width + out_x] = min_val;
    }
}

__global__ void Kernel_PoolingAverage(const uint8_t* input, uint8_t* output, int input_width, int input_height) 
{
    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;

    int out_width = input_width / 2; // TO DO: check if >> 1, which equals /2, is faster. Also try different compiler optimatization flags.
    int out_height = input_height / 2;

    if (out_x < out_width && out_y < out_height) 
    {
        int in_x = out_x * 2; // TO DO: check if << 1, which equals *2, is faster. Also try different compiler optimatization flags.
        int in_y = out_y * 2;

        int idx00 = in_y * input_width + in_x;
        int idx01 = idx00 + 1;
        int idx10 = idx00 + input_width;
        int idx11 = idx10 + 1;

        uint16_t sum = input[idx00] + input[idx01] + input[idx10] + input[idx11];
        output[out_y * out_width + out_x] = sum >> 2; // equivalent to / 4
    }
}
