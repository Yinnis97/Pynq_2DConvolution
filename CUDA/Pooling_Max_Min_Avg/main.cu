#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "BMP.h"
#include "Pooling.h"
#include "EdgeDetection_Defines.h"

#define POOL_MAX_OPERATIONS 3

// IMPORTANT: THE SOBEL OPERATOR NEEDS A GRAYSCALE IMAGE
// Convert image to grayscale:       https://www.grayscaleimage.com/#downloadtoolgray

typedef struct GPU_MemoryPointers
{
    uint8_t *d_image_in; 
	uint8_t *d_image_out; 
} GPU_MemoryPointers;

typedef struct PoolingArgs
{
    bool do_max;
    bool do_min;
    bool do_avg;
    const char* input_image_path;
    const char* output_image_paths[POOL_MAX_OPERATIONS];
    int output_count;
} PoolingArgs;


// Check if the user entered an input image path and at least one pooling operation choice and one filepath for the output image on invoking the application.
bool CPU_ParsePoolingArgs(int argc, char* argv[], PoolingArgs* args);
void CPU_InitInputImage(PoolingArgs* args, BMP *bmp);
void CPU_InitPooling(Pool_Result *pool_result, BMP *bmp);

void GPU_AllocateMemory(GPU_MemoryPointers *d_memory_pointers, BMP *bmp, Pool_Result *pool_result);
void GPU_FreeMemory(GPU_MemoryPointers *d_memory_pointers);
void GPU_CopyImageData(GPU_MemoryPointers *d_memory_pointers, BMP *bmp);

// CLI invoke command: <application_filepath(.exe)> [-max] [-min] [-avg] <input_bmp_image_path(.bmp)> <output_bmp_image_path1(.bmp)> [<output_bmp_image_path2(.bmp)>] [<output_bmp_image_path3(.bmp)>]
// Example: Pooling_Max_Min_Average_NEW.exe -max -min -avg  ./BMP_Images/lena_8bpp.bmp ./BMP_Images/lena_pool_max.bmp ./BMP_Images/lena_pool_min.bmp ./BMP_Images/lena_pool_avg.bmp
int main(int argc, char *argv[])
{
    PoolingArgs args;
    BMP bmp;
    // char* filepath_edgedetection_in; 
	// char* filepath_edgedetection_out;
    GPU_MemoryPointers d_memory_pointers;
	
    // 1. Check if the user has entered a valid application invocation command.
    if(CPU_ParsePoolingArgs(argc, argv, &args) == false)
	{
        printf("Failed to start the image pooling application. Exiting application!\r\n");
        return 1;
	}

    // 2. Initialize the CPU side of the application.
    printf("Starting image compression (pooling) application...\r\n");
    CPU_InitInputImage(&args, &bmp);
    CPU_InitPooling(&pool_result, &bmp);

    // 3. Initialize the GPU side of the application.
    GPU_AllocateMemory(&d_memory_pointers, &bmp, &pool_result);
    GPU_CopyImageData(&d_memory_pointers, &bmp);

    // 4. Initialize the pooling GPU grid, blocks and threads for the kernels to run in.
    dim3 blockSize(16, 16);
    dim3 gridSize((pool_result.width + blockSize.x - 1) / blockSize.x,
                (pool_result.height + blockSize.y - 1) / blockSize.y);

    // 5. Execute the user specified pooling operations and save the output 8bpp grayscale BMP image(s).
    if(args.do_max)
    {
        printf("Max pooling start:\r\n"); 
        Kernel_PoolingMax<<<gridSize, blockSize>>>(d_memory_pointers.d_image_in, d_memory_pointers.d_image_out, bmp.width, bmp.height);
        // Wait for the convolution kernels to complete.
        hipDeviceSynchronize();
        // Copy the GPU computated results back to the CPU
        hipMemcpy(pool_result.pooling_max, d_memory_pointers.d_image_out, pool_result.width * pool_result.height * sizeof(uint8_t), hipMemcpyDeviceToHost);
        // THE JETSON FAILS TO DISPLAY THIS BMP FORMAT WITH A COUPLE IMAGE VIEWERS THAT WERE TESTED WHILE MY WINDOWS LAPTOP DISPLAYS IT JUST FINE
        BMP_SaveGrayscaleImage(args.output_image_paths[0], pool_result.pooling_max, pool_result.width, pool_result.height); 
        printf("Max pooling completed, image saved to: %s!\r\n", args.output_image_paths[0]);
    }

    if(args.do_min)
    {
        printf("Min pooling start:\r\n");
        Kernel_PoolingMin<<<gridSize, blockSize>>>(d_memory_pointers.d_image_in, d_memory_pointers.d_image_out, bmp.width, bmp.height);
        // Wait for the convolution kernels to complete.
        hipDeviceSynchronize();
        // Copy the GPU computated results back to the CPU
        hipMemcpy(pool_result.pooling_min, d_memory_pointers.d_image_out, pool_result.width * pool_result.height* sizeof(uint8_t), hipMemcpyDeviceToHost);
        // THE JETSON FAILS TO DISPLAY THIS BMP FORMAT WITH A COUPLE IMAGE VIEWERS THAT WERE TESTED WHILE MY WINDOWS LAPTOP DISPLAYS IT JUST FINE
        BMP_SaveGrayscaleImage(args.output_image_paths[1], pool_result.pooling_min, pool_result.width, pool_result.height); 
        printf("Min pooling completed, image saved to: %s!\r\n", args.output_image_paths[1]);
    }

    if(args.do_avg)
    {
        printf("Average pooling start:\r\n");
        Kernel_PoolingAverage<<<gridSize, blockSize>>>(d_memory_pointers.d_image_in, d_memory_pointers.d_image_out, bmp.width, bmp.height);
        // Wait for the convolution kernels to complete.
        hipDeviceSynchronize();
        // Copy the GPU computated results back to the CPU
        hipMemcpy(pool_result.pooling_average, d_memory_pointers.d_image_out, pool_result.width * pool_result.height * sizeof(uint8_t), hipMemcpyDeviceToHost);
        // THE JETSON FAILS TO DISPLAY THIS BMP FORMAT WITH A COUPLE IMAGE VIEWERS THAT WERE TESTED WHILE MY WINDOWS LAPTOP DISPLAYS IT JUST FINE
        BMP_SaveGrayscaleImage(args.output_image_paths[2], pool_result.pooling_average, pool_result.width, pool_result.height); 
        printf("Average pooling completed, image saved to: %s!\r\n", args.output_image_paths[2]);
    }

    // Cleanup
    GPU_FreeMemory(&d_memory_pointers);
    
    printf("Image pooling succeeded. Exiting application!\r\n");
    return 0;
}

bool CPU_ParsePoolingArgs(int argc, char* argv[], PoolingArgs* args)
{
	if (argc < 4) 
	{
        printf("Usage: %s [-max] [-min] [-avg] <input.bmp> <out1.bmp> [<out2.bmp> ...]\n", argv[0]);
        return false;
    }

    memset(args, 0, sizeof(PoolingArgs));

    uint8_t i = 1;
    uint8_t pool_op_count = 0;

    // Parse pooling flags
    while (i < argc && argv[i][0] == '-') 
	{
        if (strcmp(argv[i], "-max") == 0) 
		{
            args->do_max = true;
        } 
		else if (strcmp(argv[i], "-min") == 0) 
		{
            args->do_min = true;
        } 
		else if (strcmp(argv[i], "-avg") == 0) 
		{
            args->do_avg = true;
        } 
		else 
		{
            printf("Unknown option: %s\n", argv[i]);
            return false;
        }
        ++i;
    }

    // Count how many pooling ops were selected
    if (args->do_max) ++pool_op_count;
    if (args->do_min) ++pool_op_count;
    if (args->do_avg) ++pool_op_count;

    if (pool_op_count == 0) 
	{
        printf("Error: You must specify at least one pooling operation (-max, -min, -avg).\n");
        return false;
    }

    // Input path
    if (i >= argc) 
	{
        printf("Error: Missing input BMP file.\n");
        return false;
    }
    args->input_image_path = argv[i++];
    
    // Output paths
    if (argc - i != pool_op_count) 
	{
        printf("Error: Expected %d output BMP paths, but got %d.\n", pool_op_count, argc - i);
        return false;
    }

    for (uint8_t j = 0; j < pool_op_count; ++j) 
	{
        args->output_image_paths[j] = argv[i++];
    }
    args->output_count = pool_op_count;

    return true;
}

void CPU_InitInputImage(PoolingArgs* args, BMP *bmp)
{
    BMP_Init(bmp, args->input_image_path);
}

void CPU_InitPooling(Pool_Result *pool_result, BMP *bmp)
{
    Pooling_Initialize(pool_result, bmp->width, bmp->height);
    printf("Pooling output width: %d, height: %d\r\n", pool_result->width, pool_result->height);
}

void GPU_AllocateMemory(GPU_MemoryPointers *d_memory_pointers, BMP *bmp, Pool_Result *pool_result)
{
    hipMalloc((void **)&(d_memory_pointers->d_image_in), bmp->width * bmp->height * sizeof(uint8_t));
    hipMalloc((void **)&(d_memory_pointers->d_image_out), pool_result->width * pool_result->height * sizeof(uint8_t));
}

void GPU_CopyImageData(GPU_MemoryPointers *d_memory_pointers, BMP *bmp)
{
    hipMemcpy(d_memory_pointers->d_image_in, bmp->grayscale_data, bmp->width * bmp->height * sizeof(uint8_t), hipMemcpyHostToDevice); // Using the 1D data => The conversion from the original 1D to 2D image data can be skipped!
}

void GPU_FreeMemory(GPU_MemoryPointers *d_memory_pointers)
{
    hipFree(d_memory_pointers->d_image_in);
	hipFree(d_memory_pointers->d_image_out); 
}
